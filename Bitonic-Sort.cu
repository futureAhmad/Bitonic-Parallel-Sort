#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void bitonic_sort(int *in, int j, int i) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int i2 = pow(2,i);
    int j2 = pow(2,j-1); 

    int step_length = i2 / j2;
    int shift = step_length / 2;
    int temp;

    if(index % step_length < shift){
        if((index / i2) % 2 == 0){
            if(in[index] > in[index+shift]){
                temp = in[index];
                in[index] = in[index+shift];
                in[index+shift] = temp; 
            }
        }
        else if(in[index] < in[index+shift]) {
            temp = in[index];
            in[index] = in[index+shift];
            in[index+shift] = temp;
        }
        __syncthreads();
    }
}
int main(void){
    int *a;
    int *d_a;
    int blocks = 2;
    int threads =8;
    int numThreadBlock = blocks * threads;
    int old=-1 ; 
    
    int limit = log2(numThreadBlock);
    
    int ch=2;
    int power = 1;
    int b =0;
    while(numThreadBlock > ch){
        power = power + 1;
        b = power;
        ch = pow(2, power);
    }
    if(limit != b ){
        old = numThreadBlock;
        numThreadBlock = pow(2,b);
        limit = b;        
        }
   
    int size = sizeof(int) * numThreadBlock;
    hipMalloc( (void**) &d_a, size);
    int newIndexeis = numThreadBlock; 
    if (old != -1)
      numThreadBlock = old;
    printf("old %d new %d\n",numThreadBlock, newIndexeis);
    a = (int*) malloc(size);
    
    srand(time(NULL));
    int i;
    int check_index_full=-1;
    
    for ( i=0;i<numThreadBlock;i++) {
      a[i] = (rand() % (15 - 1 + 5)) + 5;
      check_index_full = i;
    }
    if ( (old-1) == check_index_full ){
        for(i=check_index_full+1;i<newIndexeis;i++){
            a[i] = 1;
        }
    }
    numThreadBlock = newIndexeis;
    printf("Before\n");
    for (i=0;i<numThreadBlock;i++) {
      printf("index %d number %d\n", i, a[i]);
    }

    // host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int step=1, stage; 
    printf("\nlimit%d\n", limit);
   
    while( step <= limit){
        for(stage=1; stage<=step; stage++)
            bitonic_sort<<<blocks,threads>>>(d_a, stage, step);
        step+=1;
    }
        

    // divice to host
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    printf("-------------\nAfter\n");
    for (i=0;i<numThreadBlock;i++) {
      printf("index %d number %d\n", i, a[i]);
    }

    free(a);
    hipFree(d_a); 
}

